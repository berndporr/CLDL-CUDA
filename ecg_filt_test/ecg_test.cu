#include "hip/hip_runtime.h"

#include <chrono>

#include "cldl/Net.h"

#include <iostream>
#include <stdio.h>
#include <thread>

#define _USE_MATH_DEFINES
#include <math.h>


using namespace std; 

//The nlayers should be an integer of the total number of hidden layers required not including the input layer
const int nLayers = 2;

//Neuron array should hold the number of neurons for each layer, each array element is a
//single input 
int nNeurons[nLayers];

//setting up initial inputs
const int nInputs = 100;

double delay_line[nInputs];

int main(int argc, char* argv[]){

    std::cout<<"Made it to the Start :)\n\n";



    //Opening the .dat file and the output file
    //in the final program this should be replaced with the mic inputs
    FILE *finput = fopen("ecg50hz.dat","rt");
    FILE *foutput = fopen("ecg_filtered.dat","wt");

    //
    //
    //generating a network to be used

    //Filling Neurons_array with some arbitray numbers to test network
    //Setting the output layer to be of size 1
    nNeurons[0] = nInputs;
    nNeurons[1] = 1;

    //Filling Input array with 0s array 

    for(int i = 0; i< nInputs;i++){
	delay_line[i] = 0;

    }


    //Varifying that the pointer points to the first element of the array
    //std::cout<<"Checking that the nNeurons pointed matches the values stored:\n";
    //std::cout<<"Memmory Address and value at Address    "<<nNeurons<<":     "<<*nNeurons<<"\n\n";


    //Creating the Network 

    Net *net;
    net = new Net(nLayers,nNeurons,nInputs);


    //Initialises the network with: weights, biases and activation function
    // for Weights; W_Zeroes sets to 0 , W_Ones sets to 1 , W_random sets to a randome value
    // for Bias; B_None sets to , B_Random sets to a random value
    //for activations functions; Act_Sigmoid, Act_Tanh or Act_None
    net->initNetwork(Neuron::W_RANDOM, Neuron::B_NONE, Neuron::Act_Sigmoid);

    //Setting all intial inputs to 0
    net -> setInputs(delay_line);

    //Setting Learning Rate
    net -> setLearningRate(0.00000001);

    //Setting up a variable that allows for access to read the final output of the network
    Layer *output_layer = net -> getLayer(nLayers-1);
    Neuron *output_neuron = output_layer ->getNeuron(0);
    int number_of_outputs = output_layer ->getnNeurons();


    //Getting variable that allows for access to input layer
    Layer *input_layer = net->getLayer(0);
    Neuron *input_Neuron_0 = input_layer->getNeuron(0);
    int number_of_inputs = input_layer->getnNeurons();

    std::cout << "Number of Inputs:"<<number_of_inputs<<"\n";
    std::cout << "Number of Outputs:"<<number_of_outputs<<"\n";
    std::cout << "Number of Layers:"<<net->getnLayers()<<"\n";
    std::cout << "Number of Total Neurons:"<<net->getnNeurons()<<"\n";
    std::cout << "Neurons Array:";

    for(int i = 0;i < nLayers;i++) {
	std::cout << nNeurons[i] << ",";
    }
    std::cout << "\n";




    auto start = std::chrono::high_resolution_clock::now();


    double fs = 1000; // Hz
    double noise_f = 50; //Hz
    double norm_noise_f = noise_f / fs;

    for(int i=0;;i++) 
	{
	    //reading the input signal and generating the ref_noise
	    double input_signal;		
	    if (fscanf(finput,"%lf\n",&input_signal)<1) break;

	    double ref_noise = sin(2*M_PI*norm_noise_f*(double)i);

	    //Updating the inputs to the network
	    for(int i = (nInputs-1); i > 0;i--){
		delay_line[i] = delay_line[i-1];
	    }

	    delay_line[0] = ref_noise;
        
	    net -> setInputs(delay_line);

	    //propegating the sample forwards
	    net ->propInputs();


	    //storing output of the function and calculation error
	    double canceller = net->getOutput(0);


	    double error = input_signal - canceller;


	    //Setting the backward error and updating weights
	    net->setBackwardError(error);
	    net->propErrorBackward();
	    net->updateWeights();

	    fprintf(foutput,"%f %f %f\n",error, input_signal, canceller);
	}

    auto elapsed = std::chrono::high_resolution_clock::now() - start;

    long long microseconds_taken = std::chrono::duration_cast<std::chrono::microseconds>(
											 elapsed).count();

    
    std::cout<<"Time Taken:     "<<microseconds_taken<<"µs\n";

    fclose(finput);
    fclose(foutput);

    //fprintf(stderr,"Written the filtered ECG to 'ecg_filtered.dat'\n");


    std::cout<<"Made it to the End :)\n\n\n";
    
    



}
